#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include <mma.h>
using namespace nvcuda;

#include "matmul.cuh"

////////////////////////////////////////////////////////////////////////////////

template <typename scalar_t>
__device__ inline float _cast_out(scalar_t a) {
    return (float)a;
}
template <>
__device__ inline float _cast_out(unsigned int a) {
    return (float)a;
}
template <>
__device__ inline float _cast_out(__half a) {
    return __half2float(a);
}

template <typename scalar_t>
__device__ inline scalar_t _cast_from_accum(float a) {
    return (scalar_t)a;
}
template <>
__device__ inline float _cast_from_accum<float>(float a) {
    return a;
}
template <>
__device__ inline __half _cast_from_accum<__half>(float a) {
    return __float2half(a);
}


////////////////////////////////////////////////////////////////////////////////
// actual kernel stuff

// CUDA boilerplate, don't question it
template <bool is_sparse>
__device__ void MMA_Int4_Float( BOILERPLATE_ARGS(float), SPARSE_MASK_ARGS );

template <bool is_sparse>
__device__ void MMA_Int4_Half( BOILERPLATE_ARGS(__half), SPARSE_MASK_ARGS );

template <typename scalar_t>
__global__ void MMV_Int4_Dense( RECEIVE_BOILERPLATE_ARGS(scalar_t) );

template <typename scalar_t>
__global__ void MMV_Int4_Sparse( RECEIVE_BOILERPLATE_ARGS(scalar_t), RECEIVE_SPARSE_MASK_ARGS );

////////////////////////////////////////////////////////////////////////////////

// invalid specializations

// fp32 matmul not supported now; look for older commits (0d20a43 or older if
// you need them for some reason)
template <bool is_sparse>
__device__ void MMA_Int4_Float<is_sparse>(
    BOILERPLATE_ARGS(float),
    SPARSE_MASK_ARGS
) { assert(0); }

template <typename scalar_t>
__global__ void MMV_Int4_Dense(
    RECEIVE_BOILERPLATE_ARGS(scalar_t)
) { assert(0); }

template <typename scalar_t>
__global__ void MMV_Int4_Sparse<float>(
    RECEIVE_BOILERPLATE_ARGS(scalar_t),
    RECEIVE_SPARSE_MASK_ARGS
) { assert(0); }

////////////////////////////////////////////////////////////////////////////////

union BUF_TYPE {
    __half h[BLOCK_SIZE * BUF_MTX_WIDTH];
    float f[BLOCK_SIZE * BUF_MTX_F32_WIDTH];
};

typedef wmma::fragment<wmma::matrix_a, 16, 16, 16, __half, wmma::row_major> WMMA_A_FRAG;
typedef wmma::fragment<wmma::matrix_b, 16, 16, 16, __half, wmma::row_major> WMMA_B_FRAG;
typedef wmma::fragment<wmma::matrix_b, 16, 16, 16, __half, wmma::col_major> WMMA_B_FRAG_COL;
typedef wmma::fragment<wmma::accumulator, 16, 16, 16, float> WMMA_ACC_FRAG;

////////////////////////////////////////////////////////////////////////////////
// factored out loads

// conventions: offset is location within the block, pos is the location within the matrices

__device__ void gmem_load_multiplier(
    const __half* __restrict__ mult_in,
    __half* __restrict__ buf,

    const size_t i_offset,
    const size_t j_offset,

    const size_t i_pos,
    const size_t j_pos,

    const size_t in_size,
    const size_t seq_len
) {
    const auto off_end = mult_in + in_size*seq_len;
    const __half ZERO = __float2half(0.0f);

    // TODO: adjust these schemes for other tile sizes

#ifdef RESPECT_STRICT_ALIASING
    const size_t downset = i_pos + i_offset;

    const size_t back_increment = 4*BUF_MTX_WIDTH - WARP_SIZE;
    const size_t in_back_increment = 4*in_size - WARP_SIZE;

    const __half* m_in = mult_in + in_size*downset + j_offset + j_pos;
    __half* m_out = buf + BUF_MTX_WIDTH*i_offset + j_offset;
    #pragma unroll
    for(auto i = 0; i < (BLOCK_SIZE / 4); ++i) {
        if (m_in < off_end) {
            *m_out = *m_in;
            m_in += WARP_SIZE;
            m_out += WARP_SIZE;
        } else {
            *m_out = ZERO;
        }

        if (m_in < off_end) {
            *m_out = *m_in;
            m_in += in_back_increment;
            m_out += back_increment;
        } else {
            *m_out = ZERO;
        }
    }
#else
    // TODO: make positioning (offsetting) more consistent
    const size_t downset = i_pos + i_offset*WMMA_CHUNK_SIZE;

    const __half* m_in = mult_in + in_size*downset + j_offset*2 + j_pos;
    __half* m_out = buf + BUF_MTX_WIDTH*i_offset*WMMA_CHUNK_SIZE + j_offset*2;

    const int C_ZERO = *(int*)&ZERO;

    #pragma unroll
    for(auto i = 0; i < (BLOCK_SIZE / 4); ++i) {
        if (m_in < off_end) {
            *(int*)m_out = *(int*)m_in;
        } else {
            *(int*)m_out = C_ZERO;
        }

        m_out += BUF_MTX_WIDTH;
        m_in += in_size;
    }
#endif
}


template <bool is_transposed>
__device__ size_t smem_mm_calc_coord(
    const size_t i, const size_t j, const size_t k, const size_t warp_idx
);

template <>
__device__ size_t smem_mm_calc_coord<false>(
    const size_t i, const size_t j, const size_t k, const size_t warp_idx
) {
    return (
        (j*WMMA_CHUNK_SIZE + k*WMMA_CHUNK_SIZE*BUF_MTX_WIDTH) // sweep-based
        + (((warp_idx%SB_DIM_B)*SB_DIM_N)*WMMA_CHUNK_SIZE) // warp-based
    );
}
template <>
__device__ size_t smem_mm_calc_coord<true>(
    const size_t i, const size_t j, const size_t k, const size_t warp_idx
) {
    return (
        (j*WMMA_CHUNK_SIZE*BUF_MTX_WIDTH + k*WMMA_CHUNK_SIZE) // sweep-based
        + (((warp_idx%SB_DIM_B)*SB_DIM_N)*WMMA_CHUNK_SIZE*BUF_MTX_WIDTH) // warp-based
    );
}

template <typename a_type, typename b_type, bool is_transposed>
__device__ void gen_smem_block_matmul(
    WMMA_ACC_FRAG acc[][SB_DIM_N],
    const __half* __restrict__ mult,
    const __half* __restrict__ wts,
    const size_t warp_index
) {
    a_type a;
    b_type b[SB_DIM_N];

    #pragma unroll
    for(auto k = 0; k < SB_DIM_K; ++k) {
        // go down the multiplier dimension
        #pragma unroll
        for(auto i = 0; i < SB_DIM_M; ++i) {
            // and now: load in; this one is actually genuinely weird
            wmma::load_matrix_sync(
                a,
                (
                    mult
                    // sweep-based position
                    + (k*WMMA_CHUNK_SIZE + i*WMMA_CHUNK_SIZE*BUF_MTX_WIDTH)
                    // warp-based position
                    + (((warp_index/SB_DIM_B)*SB_DIM_M)*WMMA_CHUNK_SIZE*BUF_MTX_WIDTH)
                ),
                BUF_MTX_WIDTH
            );

            // go across the weights dimension
            #pragma unroll
            for(auto j = 0; j < SB_DIM_N; ++j) {
                if (i == 0) {
                    // first load; gets recycled of course
                    wmma::load_matrix_sync(
                        b[j],
                        (
                            wts + smem_mm_calc_coord<is_transposed>(i, j, k, warp_index)
                        ),
                        BUF_MTX_WIDTH
                    );
                }

                wmma::mma_sync(acc[i][j], a, b[j], acc[i][j]);
            }
        }

    }
}

template <bool is_sparse>
__device__ void smem_block_matmul(
    WMMA_ACC_FRAG acc[][SB_DIM_N],
    const __half* __restrict__ mult,
    const __half* __restrict__ wts,
    const size_t warp_index
);
template <>
__device__ void smem_block_matmul<false>(
    WMMA_ACC_FRAG acc[][SB_DIM_N],
    const __half* __restrict__ mult,
    const __half* __restrict__ wts,
    const size_t warp_index
) {
    gen_smem_block_matmul<WMMA_A_FRAG, WMMA_B_FRAG, false>(acc, mult, wts, warp_index);
}
template <>
__device__ void smem_block_matmul<true>(
    WMMA_ACC_FRAG acc[][SB_DIM_N],
    const __half* __restrict__ mult,
    const __half* __restrict__ wts,
    const size_t warp_index
) {
    gen_smem_block_matmul<WMMA_A_FRAG, WMMA_B_FRAG_COL, true>(acc, mult, wts, warp_index);
}


__device__ void smem_write_accs(
    WMMA_ACC_FRAG acc[][SB_DIM_N],
    float* __restrict__ write_buf,
    const size_t warp_index
) {
    #pragma unroll
    for(auto i = 0; i < SB_DIM_M; ++i) {
        // go across the weights dimension
        #pragma unroll
        for(auto j = 0; j < SB_DIM_N; ++j) {
            // L1 issues *here*, apparently...?
            wmma::store_matrix_sync(
                write_buf
                + (j + (warp_index%SB_DIM_B)*SB_DIM_N)*WMMA_CHUNK_SIZE
                + (i + (warp_index/SB_DIM_B)*SB_DIM_M)*BUF_MTX_F32_WIDTH*WMMA_CHUNK_SIZE,
                acc[i][j],
                BUF_MTX_F32_WIDTH,
                wmma::mem_row_major
            );
        }
    }
}

////////////////////////////////////////////////////////////////////////////////

// temporary specialization
template <bool is_sparse>
__device__ void MMA_Int4_Half(
    BOILERPLATE_ARGS(__half),
    SPARSE_MASK_ARGS
) {
    __shared__ BUF_TYPE base_buf;
    auto buf = base_buf.h;

    // TODO: investigate putting unpacked weights to bottom of packed buf and
    //       unpacking top-to-bottom; should save enough memory to boost warp
    //       occupancy on a 3080; stacking buffers vertically might make this
    //       easier (but cause bank conflicts (?))
    __shared__ uint32_t wt_buf[(BLOCK_SIZE / 8) * BLOCK_SIZE];

    __half* wt_unpacked_buf = buf + BLOCK_SIZE;

    const __half ZERO = __float2half(0.0f);

    const size_t warpIdx = threadIdx.y;
    // TODO: properly index these
    const size_t warpSubIdx = warpIdx % 2;
    const size_t warpSuperIdx = warpIdx / 2;
    const size_t warpOffset = warpSubIdx * WARP_SIZE + threadIdx.x;


    // adjust for batch dim
    multiplier += blockIdx.z*seq_len*in_size;
    outs += blockIdx.z*seq_len*out_size;

    WMMA_ACC_FRAG acc[SB_DIM_M][SB_DIM_N];
    #pragma unroll
    for (auto i = 0; i < SB_DIM_M; ++i) {
        #pragma unroll
        for (auto j = 0; j < SB_DIM_N; ++j) {
            wmma::fill_fragment(acc[i][j], 0.0f);
        }
    }


    // TODO: bake in group quant fetches

    for(size_t mtx_i = 0; mtx_i < mtx_in_size; mtx_i += BLOCK_SIZE/8) {
        __half scale = ZERO;
        __half zero = ZERO;
        {
            const auto j_idx = (blockIdx.x*BLOCK_SIZE + warpOffset) + ((mtx_i*8)/group_size)*out_size;
            scale = scales[j_idx];
            zero = zeros[j_idx];
            #ifdef FMA_TRANSFORM
                zero = __hneg(__hmul(zero, scale))
            #endif
        }

        // GMEM loading chunk

        // grab the weights first since there's no bounds-checking to desync things
        gmem_load_weights<is_sparse, BLOCK_SIZE>::load(
            matrix, wt_buf, sparse_mask,
            warpSuperIdx, warpOffset,
            mtx_i, blockIdx.x*BLOCK_SIZE,
            out_size);

        gmem_load_multiplier(multiplier, buf,
                             warpIdx, threadIdx.x,
                             blockIdx.y*BLOCK_SIZE, mtx_i*8,
                             in_size, seq_len);
        __syncthreads();

        // everything is now in GMEM; all our operations now touch only SMEM

        // step 1: unpack packed weights from SMEM -> SMEM
        smem_unpack_weights<is_sparse, BLOCK_SIZE, BUF_MTX_WIDTH, Quantization::DYNAMIC_EXPONENT_SYM>::load(
            wt_buf, wt_unpacked_buf,
            warpSuperIdx, warpOffset,
            scale, zero);

        // step 2: actual matrix mult
        smem_block_matmul<is_sparse>(acc, buf, wt_unpacked_buf, warpIdx);
        __syncthreads();
    }

    // we're done with the main loop; dump our data back to GMEM
    float* out_buf = base_buf.f;

    // dump accumulators into our write buffer
    smem_write_accs(acc, out_buf, warpIdx);

    __syncthreads(); // be very certain

    // and then dump from SMEM => GMEM
    #ifdef RESPECT_STRICT_ALIASING
        #pragma unroll
        for(auto i = 0; i < (BLOCK_SIZE / 2); ++i) {
            const size_t write_j = blockIdx.x*BLOCK_SIZE + warpOffset;
            const size_t write_i = blockIdx.y*BLOCK_SIZE + i*2 + warpSuperIdx;
            if ((write_j < out_size) && (write_i < seq_len)) {
                outs[write_j + write_i*out_size] = \
                    __float2half(out_buf[warpOffset + (warpSuperIdx + i*2)*BUF_MTX_F32_WIDTH]);
            }
        }
    #else
        half a[2]; // temp buf because i don't trust half2s
        #pragma unroll
        for(auto i = 0; i < (BLOCK_SIZE / 4); ++i) {
            const size_t write_j = blockIdx.x*BLOCK_SIZE + threadIdx.x*2;
            const size_t write_i = blockIdx.y*BLOCK_SIZE + warpIdx*WMMA_CHUNK_SIZE + i;
            const size_t v = warpIdx*WMMA_CHUNK_SIZE + i;
            if ((write_j < out_size) && (write_i < seq_len)) {
                a[0] = __float2half(out_buf[threadIdx.x*2 + v*BUF_MTX_F32_WIDTH]);
                a[1] = __float2half(out_buf[threadIdx.x*2+1 + v*BUF_MTX_F32_WIDTH]);

                *(int*)(outs + write_j + write_i*out_size) = *(int*)&a;
            }
        }
    #endif
}

////////////////////////////////////////////////////////////////////////////////

template <>
__global__ void MMV_Int4_Dense<float>(
    RECEIVE_BOILERPLATE_ARGS(float)
) {
    MMA_Int4_Float<false>(
        outs, reinterpret_cast<const uint32_t*>(matrix), multiplier, scales, zeros,
        group_size, in_size, seq_len, mtx_in_size, out_size, nullptr
    );
}
template <>
__global__ void MMV_Int4_Sparse<float>(
    RECEIVE_BOILERPLATE_ARGS(float),
    RECEIVE_SPARSE_MASK_ARGS
) {
    MMA_Int4_Float<true>(
        outs, reinterpret_cast<const uint32_t*>(matrix), multiplier, scales, zeros,
        group_size, in_size, seq_len, mtx_in_size, out_size, reinterpret_cast<const uint32_t*>(sparse_mask)
    );
}

template <>
__global__ void MMV_Int4_Dense<c10::Half>(
    RECEIVE_BOILERPLATE_ARGS(c10::Half)
) {
    MMA_Int4_Half<false>(
        // torch stuff; torch doesn't store halfs internally as CUDA halfs,
        // but they *are* bit-compatible so reinterpret_cast solves the issue
        reinterpret_cast<__half*>(outs),
        reinterpret_cast<const uint32_t*>(matrix),
        reinterpret_cast<const __half*>(multiplier),
        reinterpret_cast<const __half*>(scales),
        reinterpret_cast<const __half*>(zeros),
        group_size, in_size, seq_len, mtx_in_size, out_size, nullptr
    );
}
template <>
__global__ void MMV_Int4_Sparse<c10::Half>(
    RECEIVE_BOILERPLATE_ARGS(c10::Half),
    RECEIVE_SPARSE_MASK_ARGS
) {
    MMA_Int4_Half<true>(
        reinterpret_cast<__half*>(outs),
        reinterpret_cast<const uint32_t*>(matrix),
        reinterpret_cast<const __half*>(multiplier),
        reinterpret_cast<const __half*>(scales),
        reinterpret_cast<const __half*>(zeros),
        group_size, in_size, seq_len, mtx_in_size, out_size,
        reinterpret_cast<const uint32_t*>(sparse_mask)
    );
}


void matmul_int4(
    torch::Tensor outs,
    torch::Tensor matrix,
    torch::Tensor x,
    torch::Tensor scales,
    torch::Tensor zeros,
    int group_size,
    c10::optional<torch::Tensor> sparse_mask
) {
    const bool is_sparse = sparse_mask.has_value() && sparse_mask.value().defined();

    if (group_size < 0) {
        // don't pick something too big in case of weird uint shenanigans
        group_size = 0x0FFFFFFF;
    } else {
        assert(group_size % 64 == 0);
    }

    // perform matrix multiplication:
    //    x * W^T = O
    // x : [batch_size, seq_len, in_size]
    // W : [out_size, in_size]
    // O : [batch_size, seq_len, out_size]
    //
    // with zeros and scales such that
    //    W = scales * Wq + zeros
    // scales : [n_groups, out_size,]
    // zeros  : [n_groups, out_size,]

    const auto batch_size = x.size(0);
    const auto seq_len = x.size(1);
    const auto in_size = x.size(2);

    const auto mtx_in_size = matrix.size(0);
    const auto out_size = matrix.size(1);

    if (is_sparse) {
        assert((mtx_in_size*16) == in_size);
    } else {
        assert((mtx_in_size*8) == in_size);
    }
    assert(outs.size(0) == batch_size);
    assert(outs.size(1) == seq_len);
    assert(outs.size(2) == out_size);
    assert(zeros.size(1) == out_size);
    assert(scales.size(1) == out_size);
    assert((group_size * zeros.size(0)) >= in_size);
    assert((group_size * scales.size(0)) >= in_size);
    assert(out_size % BLOCK_SIZE == 0);

    const auto THREAD_X = WARP_SIZE;
    const auto THREAD_Y = WMMA_CHUNK_COUNT;

    dim3 threads(THREAD_X, THREAD_Y);
    dim3 blocks(
        (out_size + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (seq_len + BLOCK_SIZE - 1) / BLOCK_SIZE,
        batch_size
    );

    if (is_sparse) {
        auto actual_sparse = sparse_mask.value();
        assert(actual_sparse.size(0)*2 == mtx_in_size);
        assert(actual_sparse.size(1) == out_size);

        AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.type(), "mmv_int4", ([&] {
            MMV_Int4_Sparse<<<blocks, threads>>>(
                outs.data<scalar_t>(),
                matrix.data<int32_t>(),
                x.data<scalar_t>(),
                scales.data<scalar_t>(),
                zeros.data<scalar_t>(),
                // multiply mtx_in_size by 2 to pretend it's still in terms of
                // weights; easier indexing
                group_size, in_size, seq_len, mtx_in_size*2, out_size, actual_sparse.data<int32_t>()
            );
        }));
    } else {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.type(), "mmv_int4", ([&] {
            MMV_Int4_Dense<<<blocks, threads>>>(
                outs.data<scalar_t>(),
                matrix.data<int32_t>(),
                x.data<scalar_t>(),
                scales.data<scalar_t>(),
                zeros.data<scalar_t>(),
                group_size, in_size, seq_len, mtx_in_size, out_size
            );
        }));
    }
}
